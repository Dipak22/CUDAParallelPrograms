
#include <hip/hip_runtime.h>
#include<iostream>
#define LENGTH 2048
using namespace std;

__global__ void sumReduce(int *arr, int *out){
   __shared__ int arr_s[LENGTH/2];

   int idx = threadIdx.x;
   arr_s[idx] = arr[idx] + arr[idx + blockDim.x];
   for(int stride = blockDim.x/2; stride>=1; stride /=2){
        __syncthreads();
       if(threadIdx.x<stride){
           arr_s[idx] +=arr_s[idx+stride];
       }
   }
   if(threadIdx.x==0)
       out[0] = arr_s[0];
}

int main(){
   int *arr, *sum;
   arr = (int*)malloc(LENGTH * sizeof(int));
   sum = (int*)malloc(sizeof(int));
   for(int i=0;i<LENGTH;i++)
       arr[i] = i+1;
   int *arr_d, *output;
   hipMalloc((void**)&arr_d, LENGTH * sizeof(int));
   hipMalloc((void**)&output, sizeof(int));
   hipMemcpy(arr_d, arr, LENGTH*sizeof(int), hipMemcpyHostToDevice);
   dim3 dimBlock(1024);
   dim3 dimGrid(1);
   sumReduce<<<dimGrid,dimBlock>>>(arr_d, output);
   hipMemcpy(sum, output, sizeof(int), hipMemcpyDeviceToHost);
   cout<<sum[0];

   hipFree(arr_d);
   hipFree(output);

   free(arr);
   free(sum);

   return 0;
}
 
#include <hip/hip_runtime.h>
#include<iostream>
 #define LENGTH 2048
 using namespace std;

 __global__ void sumReduce(int *arr, int *out){
    int idx = 2*threadIdx.x;
    for(int stride = 1; stride<=blockDim.x; stride *=2){
        if(threadIdx.x%stride ==0){
            arr[idx] +=arr[idx+stride];
        }
        __syncthreads();
    }
    if(threadIdx.x==0)
        out[0] = arr[0];
 }

 int main(){
    int *arr, *sum;
    arr = (int*)malloc(LENGTH * sizeof(int));
    sum = (int*)malloc(sizeof(int));
    for(int i=0;i<LENGTH;i++)
        arr[i] = i+1;
    int *arr_d, *output;
    hipMalloc((void**)&arr_d, LENGTH * sizeof(int));
    hipMalloc((void**)&output, sizeof(int));
    hipMemcpy(arr_d, arr, LENGTH*sizeof(int), hipMemcpyHostToDevice);
    dim3 dimBlock(1024);
    dim3 dimGrid(1);
    sumReduce<<<dimGrid,dimBlock>>>(arr_d, output);
    hipMemcpy(sum, output, sizeof(int), hipMemcpyDeviceToHost);
    cout<<sum[0];

    hipFree(arr_d);
    hipFree(output);

    free(arr);
    free(sum);

    return 0;
 }

#include <hip/hip_runtime.h>
#include<iostream>
#define LENGTH 4096
#define BLOCKDIM 512
#define COARSE_FACTOR 2
using namespace std;

__global__ void sumReduce(int *arr, int *out){
   __shared__ int arr_s[BLOCKDIM];
   int segment = COARSE_FACTOR* 2 * blockDim.x * blockIdx.x;
   int idx = threadIdx.x;
   int i = segment + idx;
   float sum = arr[i];
   for(int tile = 1; tile<COARSE_FACTOR*2;tile++)
        sum +=arr[i + tile*BLOCKDIM];
   arr_s[idx] = sum;
   for(int stride = blockDim.x/2; stride>=1; stride /=2){
        __syncthreads();
       if(threadIdx.x<stride){
           arr_s[idx] +=arr_s[idx+stride];
       }
   }
   if(threadIdx.x==0)
       atomicAdd(&out[0], arr_s[0]);
}

int main(){
   int *arr, *sum;
   arr = (int*)malloc(LENGTH * sizeof(int));
   sum = (int*)malloc(sizeof(int));
   for(int i=0;i<LENGTH;i++)
       arr[i] = i+1;
   int *arr_d, *output;
   hipMalloc((void**)&arr_d, LENGTH * sizeof(int));
   hipMalloc((void**)&output, sizeof(int));
   hipMemcpy(arr_d, arr, LENGTH*sizeof(int), hipMemcpyHostToDevice);
   dim3 dimBlock(BLOCKDIM);
   dim3 dimGrid((LENGTH + BLOCKDIM -1)/BLOCKDIM);
   sumReduce<<<dimGrid,dimBlock>>>(arr_d, output);
   hipMemcpy(sum, output, sizeof(int), hipMemcpyDeviceToHost);
   cout<<sum[0];

   hipFree(arr_d);
   hipFree(output);

   free(arr);
   free(sum);

   return 0;
}